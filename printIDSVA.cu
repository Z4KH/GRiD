// nvcc -gencode arch=compute_89,code=sm_89 -o printIDSVA printIDSVA.cu


#include <random>
#include <algorithm>
#include "grid.cuh"
#define RANDOM_MEAN 0
#define RANDOM_STDEV 1
std::default_random_engine randEng(1337); // fixed seed
std::normal_distribution<double> randDist(RANDOM_MEAN, RANDOM_STDEV); //mean followed by stdiv
template <typename T>
T getRand(){return static_cast<T>(randDist(randEng));}








template <typename T>
__host__
void test(){
	T gravity = static_cast<T>(9.81);
	dim3 dimms(grid::SUGGESTED_THREADS,1,1);
	
	hipStream_t *streams = grid::init_grid<T>();
	
	grid::robotModel<T> *d_robotModel = grid::init_robotModel<T>();
	
	grid::gridData<T> *hd_data = grid::init_gridData<T,1>();
	
	
	
	T q[] = {0.300623, -1.427442, 0.047334, -0.51204, -1.437442, 0.500384, -0.881586};
	T qd[] = {-1.226503, -0.619695, 0.973148, -0.750689, -0.253769, 0.493305, -0.695605};
	T qdd[] = {0.425334, 0.340006, -0.178834, -0.013169, -2.349815, 0.405039, -2.266609};
	// T q[] = {0.300623, -1.427442, 0.047334, -0.51204, -1.437442, 0.500384, -0.881586, -1.226503, -0.619695, 0.973148, -0.750689, -0.253769};
	// T qd[] = {0.493305, -0.695605, 0.425334, 0.340006, -0.178834, -0.013169, -2.349815, 0.405039, -2.266609, -0.424634, 1.034167, -0.270165};
	// T qdd[] = {0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0};
	
	// load q,qd,u
	for(int j = 0; j < grid::NUM_JOINTS; j++){
		// hd_data->h_q_qd_u[j] = getRand<double>(); 
		// hd_data->h_q_qd_u[j+grid::NUM_JOINTS] = getRand<double>(); 
		// hd_data->h_q_qd_u[j+2*grid::NUM_JOINTS] = getRand<double>();
		hd_data->h_q_qd_u[j] = q[j];; 
		hd_data->h_q_qd_u[j+grid::NUM_JOINTS] = qd[j];; 
		hd_data->h_q_qd_u[j+2*grid::NUM_JOINTS] = qdd[j];
	}
	gpuErrchk(hipMemcpy(hd_data->d_q_qd_u,hd_data->h_q_qd_u,3*grid::NUM_JOINTS*sizeof(T),hipMemcpyHostToDevice));
	gpuErrchk(hipDeviceSynchronize());

	grid::inverse_dynamics_single_timing<T>(hd_data,d_robotModel,gravity,1,dim3(1,1,1),dimms,streams);
	grid::inverse_dynamics_single_timing<T>(hd_data,d_robotModel,gravity,1,dim3(1,1,1),dimms,streams);
	grid::inverse_dynamics_gradient_single_timing<T>(hd_data,d_robotModel,gravity,1,dim3(1,1,1),dimms,streams);
	grid::idsva_so_host_single_timing<T>(hd_data,d_robotModel,gravity,1,dim3(1,1,1),dimms,streams);
	
	// // Print Results
	printf("d2tau_dq2\n");
	for (int i = 0; i < grid::NUM_JOINTS; i++){
		printf("Joint %i\n",i);
		printMat<T,grid::NUM_JOINTS,grid::NUM_JOINTS>(&hd_data->h_idsva_so[i*grid::NUM_JOINTS*grid::NUM_JOINTS],grid::NUM_JOINTS);
		printf("\n\n");
	}
	printf("\n\n\n\nd2tau_dqd2\n");
	for (int i = 0; i < grid::NUM_JOINTS; i++){
		printf("Joint %i\n",i);
		printMat<T,grid::NUM_JOINTS,grid::NUM_JOINTS>(&hd_data->h_idsva_so[grid::NUM_JOINTS*grid::NUM_JOINTS*grid::NUM_JOINTS+i*grid::NUM_JOINTS*grid::NUM_JOINTS],grid::NUM_JOINTS);
		printf("\n\n");
	}
	printf("\n\n\n\nd2tau_cross\n");
	for (int i = 0; i < grid::NUM_JOINTS; i++){
		printf("Joint %i\n",i);
		printMat<T,grid::NUM_JOINTS,grid::NUM_JOINTS>(&hd_data->h_idsva_so[2*grid::NUM_JOINTS*grid::NUM_JOINTS*grid::NUM_JOINTS+i*grid::NUM_JOINTS*grid::NUM_JOINTS],grid::NUM_JOINTS);
		printf("\n\n");
	}
	printf("\n\n\n\ndM_dq\n");
	for (int i = 0; i < grid::NUM_JOINTS; i++){
		printf("Joint %i\n",i);
		printMat<T,grid::NUM_JOINTS,grid::NUM_JOINTS>(&hd_data->h_idsva_so[3*grid::NUM_JOINTS*grid::NUM_JOINTS*grid::NUM_JOINTS+i*grid::NUM_JOINTS*grid::NUM_JOINTS],grid::NUM_JOINTS);
		printf("\n\n");
	}


	grid::close_grid<T>(streams,d_robotModel,hd_data);
}

int main(void){
	test<float>(); 
	return 0;
}
